#include "hip/hip_runtime.h"
/****************************************************************************
** Copyright (c) 2012 Hugues Bruant <hugues@cmu.edu>
** All rights reserved.
**
** This file may be used under the terms of the GNU General Public License
** version 3 as published by the Free Software Foundation.
**
** This file is provided AS IS with NO WARRANTY OF ANY KIND, INCLUDING THE
** WARRANTY OF DESIGN, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.
****************************************************************************/

#include "casim.h"
#include "cycleTimer.h"

#include <algorithm>
#include <cmath>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

struct CARule {
    enum Type {
        LifeLike,
        WireWorld
    };
    
    CARule(const char *s);
    const char* toString() const;
    
    unsigned short type;
    uint16_t B, S; // parameters for Life-like
};

struct CAParams {
    unsigned short type;
    uint16_t B, S;
    
    unsigned int width;
    unsigned int height;
    unsigned int *workOffset;
};

__constant__ CAParams caParams;

////////////////////////////////////////////////////////////////////////////////

enum {
    kMacroCellWidth = 32,
    kMacroCellHeight = 16
};

// core CA update
// responsible for updating 4 adjacent cells in a rows
// * mid is the old values of the 4 cells to update (little endian)
// * sp is the shared memory array containing the old values of all cells in the
// macrocell being updated plus all boundary cells (Moore neighbourhood). Aligned
// on 4-byte boundary
// * return the new values of the cells (same encoding)
template <uint16_t T>
inline __device__ uint32_t CAUpdateCore(uint32_t mid, uint8_t *sp) {
    uint32_t nval = 0;
    if (T == CARule::WireWorld) {
        // WireWorld
        // TODO:
        
    } else {
        // Life-like
        const int S = caParams.S, B = caParams.B;
        
        const int stride = kMacroCellWidth+8;
        // NOTE: assume GPU in little-endian mode
        // TODO: get driver to ensure little-endian mode during setup
        const uint32_t top = *reinterpret_cast<uint32_t*>(sp-stride);
        const uint32_t bot = *reinterpret_cast<uint32_t*>(sp+stride);
        
        const int atop = (top & 0xff), btop = ((top >> 8) & 0xff), ctop = ((top >> 16) & 0xff), dtop  = ((top >> 24) & 0xff);
        const int amid = (mid & 0xff), bmid = ((mid >> 8) & 0xff), cmid = ((mid >> 16) & 0xff), dmid  = ((mid >> 24) & 0xff);
        const int abot = (bot & 0xff), bbot = ((bot >> 8) & 0xff), cbot = ((bot >> 16) & 0xff), dbot  = ((bot >> 24) & 0xff);
        
        int sum =
            sp[-stride-1] + atop + btop +
            sp[       -1]        + bmid +
            sp[ stride-1] + abot + bbot;
        
        nval |= ((1 << sum) & (amid ? S : B)) ? 1 : 0;
        
        sum =
            atop + btop + ctop +
            amid        + cmid +
            abot + bbot + cbot;
        
        nval |= ((1 << sum) & (bmid ? S : B)) ? (1 << 8) : 0;
        
        sum =
            btop + ctop + dtop +
            bmid        + dmid +
            bbot + cbot + dbot;
        
        nval |= ((1 << sum) & (cmid ? S : B)) ? (1 << 16) : 0;
        
        sum =
            ctop + dtop + sp[-stride+4] +
            cmid        + sp[        4] +
            cbot + dbot + sp[ stride+4];
        
        nval |= ((1 << sum) & (dmid ? S : B)) ? (1 << 24) : 0;
    }
    return nval;
}

//
//
inline __device__ uint32_t loadCells(int row, int col, uint8_t *sp, uint8_t *ip, size_t pitch) {
    uint32_t mid;
    
    // load inner cells cooperatively
    if (row < caParams.height && col < caParams.width) {
        mid = *reinterpret_cast<uint32_t*>(ip);
        *reinterpret_cast<uint32_t*>(sp) = mid;
        
        // load left and right boundary cells
        if (threadIdx.x == 0) {
            sp[-1] = blockIdx.x == 0 ? 0 : ip[-1];
        } else if (threadIdx.x == blockDim.x-1) {
            sp[4] = blockIdx.x == gridDim.x-1 ? 0 : ip[4];
        }
        
        // load top and bottom boundary cells
        if (threadIdx.y == 0) {
            sp -= (kMacroCellWidth+8);
            ip -= pitch;
            *reinterpret_cast<uint32_t*>(sp) = *reinterpret_cast<uint32_t*>(ip);
            if (threadIdx.x == 0) {
                sp[-1] = blockIdx.x == 0 ? 0 : ip[-1];
            } else if (threadIdx.x == blockDim.x-1) {
                sp[4] = blockIdx.x == gridDim.x-1 ? 0 : ip[4];
            }
            sp += (kMacroCellWidth+8);
        } else if (threadIdx.y == blockDim.y-1) {
            sp += (kMacroCellWidth+8);
            ip += pitch;
            *reinterpret_cast<uint32_t*>(sp) = *reinterpret_cast<uint32_t*>(ip);
            if (threadIdx.x == 0) {
                sp[-1] = blockIdx.x == 0 ? 0 : ip[-1];
            } else if (threadIdx.x == blockDim.x-1) {
                sp[4] = blockIdx.x == gridDim.x-1 ? 0 : ip[4];
            }
            sp -= (kMacroCellWidth+8);
        }
    } else {
        *reinterpret_cast<uint32_t*>(sp) = 0;
        
        // load left and right boundary cells
        if (threadIdx.x == 0) {
            sp[-1] = 0;
        } else if (threadIdx.x == blockDim.x-1) {
            sp[4] = 0;
        }
    }
    return mid;
}

// naive, embarassingly parallel CA update
template <uint16_t T>
__global__ void kernelCAUpdateNaive(uint8_t *in,  uint8_t *out, size_t pitch) {
    __shared__ uint8_t ocells[(kMacroCellHeight+2)*(kMacroCellWidth+8)];
    
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    uint8_t *ip = in + row * pitch + col * 4;
    
    // cooperative load of relevant old cell values to shared memory
    uint8_t *sp = ocells + (threadIdx.y+1) * (kMacroCellWidth+8) + (threadIdx.x+1) * 4;
    
    uint32_t mid = loadCells(row, col, sp, ip, pitch);
    
    // wait for shared array to be fully initialized
    __syncthreads();
    
    // only update relevant cells
    if (row >= caParams.height || col >= caParams.width)
        return;
    
    uint32_t nval = CAUpdateCore<T>(mid, sp);
    uint8_t *op = out + row * pitch + col * 4;
    *reinterpret_cast<uint32_t*>(op) = nval;
}

// fill initial worklist (all cells to be updated)
__global__ void kernelInitWorklist(uint32_t *work, size_t n) {
    const unsigned int w = caParams.width;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n)
        return;
    work[2*idx+0] = (idx / w) * kMacroCellHeight;
    work[2*idx+1] = (idx % w) * kMacroCellWidth;
}

// more sophisticated worklist-based approach
// may or may not be faster depending on the state
template <uint16_t T>
__global__ void kernelCAUpdateWorkList(uint32_t *iwork, uint32_t *owork,
                                       uint8_t *in, uint8_t *out, size_t pitch) {
    __shared__ unsigned int update[9];
    __shared__ uint8_t ocells[(kMacroCellHeight+2)*(kMacroCellWidth+8)];
    
    // derive row/col from worklist
    unsigned int idx = blockIdx.x;
    
    uint32_t row = iwork[2*idx+0] + threadIdx.y;
    uint32_t col = iwork[2*idx+1] + threadIdx.x;
    
    uint8_t *ip = in + row * pitch + col * 4;
    
    // cooperative load of relevant old cell values to shared memory
    uint8_t *sp = ocells + (threadIdx.y+1) * (kMacroCellWidth+8) + (threadIdx.x+1) * 4;
    
    uint32_t mid = loadCells(row, col, sp, ip, pitch);
    
    // wait for shared array to be fully initialized
    __syncthreads();
    
    if (row >= caParams.height || col >= caParams.width)
        return;
    
    uint32_t nval = CAUpdateCore<T>(mid, sp);
    uint8_t *op = out + row * pitch + col * 4;
    *reinterpret_cast<uint32_t*>(op) = nval;
    
    // determine modification
    uint32_t mod = mid ^ nval;
    
    unsigned int tidx = threadIdx.y * blockIdx.x + threadIdx.x;
    if (tidx < 9)
        update[tidx] = 0;
    
    __syncthreads();
    
    if (mod) {
        // only one thread will successfully write, doesn't matter which one
        update[4] = 1;
        
        if (blockIdx.x != 0 && threadIdx.x == 0 && (mod & 0xff)) {
            update[3] = 1;
            if (blockIdx.y != 0 && threadIdx.x == 0)
                update[0] = 1;
            if (blockIdx.y != gridDim.y-1 && threadIdx.x == blockDim.x-1)
                update[6] = 1;
        } else if (blockIdx.x != gridDim.x-1 && threadIdx.x == blockDim.x-1 && (mod & 0xff000000)) {
            update[5] = 1;
            if (blockIdx.y != 0 && threadIdx.y == 0)
                update[2] = 1;
            if (blockIdx.y != gridDim.y-1 && threadIdx.y == blockDim.x-1)
                update[8] = 1;
        }
        
        if (blockIdx.y != 0 && threadIdx.y == 0) {
            update[1] = 1;
        } else if (blockIdx.y != gridDim.y-1 && threadIdx.y == blockDim.y-1) {
            update[7] = 1;
        }
    }
    
    __syncthreads();
    
    if (threadIdx.x == 0) {
        unsigned int count = update[0] + update[1] + update[2] +
                             update[3] + update[4] + update[5] + 
                             update[6] + update[7] + update[8];
        
        unsigned int offset = atomicAdd(caParams.workOffset, count);
//         uint32_t *ow = owork + 2*offset;
//         
//         if (update[0]) { ow[0] = row-1; ow[1] = col-1; ow += 2; }
//         if (update[1]) { ow[0] = row-1; ow[1] = col;   ow += 2; }
//         if (update[2]) { ow[0] = row-1; ow[1] = col+1; ow += 2; }
//         if (update[3]) { ow[0] = row;   ow[1] = col-1; ow += 2; }
//         if (update[4]) { ow[0] = row;   ow[1] = col  ; ow += 2; }
//         if (update[5]) { ow[0] = row;   ow[1] = col;   ow += 2; }
//         if (update[6]) { ow[0] = row+1; ow[1] = col-1; ow += 2; }
//         if (update[7]) { ow[0] = row+1; ow[1] = col;   ow += 2; }
//         if (update[8]) { ow[0] = row+1; ow[1] = col+1; }
    }
}

////////////////////////////////////////////////////////////////////////////////

CARule::CARule(const char *s) {
    bool ok = false;
    if (!strcmp(s, "wire")) {
        type = WireWorld;
        ok = true;
    } else if (s[0] == 'B') {
        // B[1-8]+S[0-8]+
        type = LifeLike;
        B = 0;
        S = 0;
        int i = 1;
        while (s[i] >= '1' && s[i] <= '8')
            B |= (1 << (s[i++] - '0'));
        if (s[i] == 'S') {
            ++i;
            while (s[i] >= '1' && s[i] <= '8')
                S |= (1 << (s[i++] - '0'));
            ok = s[i] == '\0';
        }
    }
    
    if (!ok) {
        fprintf(stderr, "Invalid rule, falling back to Life\n");
        type = LifeLike;
        B = 1 << 3;
        S = (1 << 2) | (1 << 3);
    }
}

const char* CARule::toString() const {
    static char buffer[20];
    if (type == WireWorld)
        return "wire";
    char *p = buffer;
    *p++ = 'B';
    for (int i = 1; i <= 8; ++i)
        if (B & (1 << i))
            *p++ = '0' + i;
    *p++ = 'S';
    for (int i = 0; i <= 8; ++i)
        if (S & (1 << i))
            *p++ = '0' + i;
    *p = '\0';
    return buffer;
}

////////////////////////////////////////////////////////////////////////////////

CASim::CASim(const char *rule) {
    this->rule = new CARule(rule);
    
    generation = 0;
    width = height = 0;
    cell0 = cell1 = 0;
    work0 = work1 = 0;
    workOffset = 0;
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Initializing CUDA for CASim\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}

CASim::~CASim() {
    if (cell0) hipFree(cell0);
    if (cell1) hipFree(cell1);
    if (work0) hipFree(work0);
    if (work1) hipFree(work1);
    if (workOffset) hipFree(workOffset);
}

void CASim::step(int n) {
    fprintf(stderr, "Running %i steps of %s\n", n, rule->toString());
    double ref = CycleTimer::currentSeconds();
    
    for (int i = 0; i < n; ++i) {
        uint8_t *scell = (generation & 1) ? cell1 : cell0;
        uint8_t *dcell = (generation & 1) ? cell0 : cell1;
        
#if 0
        dim3 updateBlockDim(kMacroCellWidth / 4, kMacroCellHeight, 1);
        dim3 updateGridDim((width / 4 + updateBlockDim.x - 1) / updateBlockDim.x,
                           (height + updateBlockDim.y - 1) / updateBlockDim.y);
        
        if (rule->type == CARule::WireWorld)
            kernelCAUpdateNaive<CARule::WireWorld><<<updateGridDim, updateBlockDim>>>(
                scell + pitch, dcell + pitch, pitch);
        else
            kernelCAUpdateNaive<CARule::LifeLike><<<updateGridDim, updateBlockDim>>>(
                scell + pitch, dcell + pitch, pitch);
        
#else
        // when worklist contains all the macrocells, the update phase alone
        // is more than twice as slow as the naive approach as it needs to read
        // from the worklist
        // TODO: switch back and forth between naive and worklist depending on
        // the size of the worklist?
        uint32_t wAmount, wNext;
        hipMemcpy(&wAmount, workOffset, sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        
        fprintf(stderr, "updating %u macrocells\n", wAmount);
        
        hipMemset(workOffset, 0, sizeof(uint32_t));
        hipDeviceSynchronize();
        
        hipMemcpy(&wNext, workOffset, sizeof(uint32_t), hipMemcpyDeviceToHost);
        fprintf(stderr, "%u\n", wNext);
        
        dim3 updateBlockDim(kMacroCellWidth / 4, kMacroCellHeight, 1);
        
        int wLim = 1024;
        
        for (int i = 0; i < (wAmount + wLim-1)/wLim; ++i) {
            dim3 updateGridDim(min(wLim, wAmount - i*wLim), 1);
            if (rule->type == CARule::WireWorld)
                kernelCAUpdateWorkList<CARule::WireWorld><<<updateGridDim, updateBlockDim>>>(
                    work0 + i*2*wLim, work1, scell + pitch, dcell + pitch, pitch);
            else
                kernelCAUpdateWorkList<CARule::LifeLike><<<updateGridDim, updateBlockDim>>>(
                    work0 + i*2*wLim, work1, scell + pitch, dcell + pitch, pitch);
            
        }
        hipDeviceSynchronize();
        
        hipMemcpy(&wNext, workOffset, sizeof(uint32_t), hipMemcpyDeviceToHost);
        fprintf(stderr, "%u macrocells scheduled for update\n", wNext);
        
        // TODO: sort work1
//         thrust::device_ptr<uint64_t> wit(reinterpret_cast<uint64_t*>(work1));
//         thrust::stable_sort(wit, wit + wAmount);
        
        // TODO: filter work0 into work1
        
#endif
        ++generation;
        hipDeviceSynchronize();
    }
    
    double end = CycleTimer::currentSeconds();
    fprintf(stderr, "Elapsed : %lf ms (%lf / step)\n",
            (end - ref) * 1000.0, ((end - ref) * 1000.0) / (double)n);
}

bool CASim::setCells(unsigned int width, unsigned int height, uint8_t max,
                     const uint8_t *cells) {
    // check that the input respects the maximum number of states of the CA
    uint8_t maxState = rule->type == CARule::WireWorld ? 3 : 1;
    if (max > maxState) {
        fprintf(stderr, "Input max value outside of CA bounds (%u > %u).\n",
                (unsigned int)max, (unsigned int)maxState);
        return false;
    }
    
    if (cell0) hipFree(cell0);
    if (cell1) hipFree(cell1);
    if (work0) hipFree(work0);
    if (work1) hipFree(work1);
    
    double tmp, ref = CycleTimer::currentSeconds();
    
    this->width = width;
    this->height = height;
    
    generation = 0;
    
    hipError_t err;
    
    // Alloc double buffers
    // buffers are padded as follows to simplify kernels :
    // * add a top row (always set to 0)
    // * add a bottom row (always set to 0)
    // * ensure the width of the allocated array is a multiple of 16
    int wpad = (width & 15);
    if (wpad)
        wpad = 16 - wpad;
    size_t pitch0, pitch1;
    err = hipMallocPitch(&cell0, &pitch0, width + wpad, height+2);
    if (err) { fprintf(stderr, "Unable to allocate buffer\n"); return false; }
    err = hipMallocPitch(&cell1, &pitch1, width + wpad, height+2);
    if (err) { fprintf(stderr, "Unable to allocate buffer\n"); return false; }
    
    // alloc worklists
    uint32_t wh = ((width + 15) / 16) * ((height + 15) / 16);
    err = hipMalloc(&work0, wh * 2 * sizeof(uint32_t));
    if (err) { fprintf(stderr, "Unable to allocate worklist\n"); return false; }
    err = hipMalloc(&work1, wh * 9 * 2 * sizeof(uint32_t));
    if (err) { fprintf(stderr, "Unable to allocate worklist\n"); return false; }
    if (!workOffset)
        hipMalloc(&workOffset, sizeof(uint32_t));
    
    tmp = CycleTimer::currentSeconds();
    fprintf(stderr, "Alloc buffers : %lf ms\n", (tmp - ref)*1000.0);
    
    if (pitch0 != pitch1) {
        fprintf(stderr, "Pitch mismatch\n");
        return false;
    }
    pitch = pitch0;
    
    // set global CA params
    CAParams params;
    params.type = rule->type;
    params.B = rule->B;
    params.S = rule->S;
    params.width = width;
    params.height = height;
    params.workOffset = workOffset;
    hipMemcpyToSymbol(HIP_SYMBOL(caParams), &params, sizeof(CAParams));
    
    // copy initial state to first buffer
    // for now simple 1:1 mapping, each byte being one cell but may change in
    // the future to allow further optimization
    hipMemcpy2D(cell0+pitch, pitch, cells, width, width, height, hipMemcpyHostToDevice);
    
    // reset padding rows
    hipMemset(cell0, 0, width);
    hipMemset(cell0+(height+1)*pitch, 0, width);
    hipMemset(cell1, 0, width);
    hipMemset(cell1+(height+1)*pitch, 0, width);
    
    // reset padding columns
    if (wpad) {
//         for (int i = 0; i < height+2; ++i) {
//             hipMemset(cell0+i*pitch+width, 0, wpad);
//             hipMemset(cell1+i*pitch+width, 0, wpad);
//         }
        hipMemset2D(cell0+width, pitch, 0, wpad, height+2);
    }
    
    
    // prepare initial worklist
    dim3 blockDim(128, 1);
    dim3 gridDim((wh + 127) / 128, 1);
    kernelInitWorklist<<<gridDim, blockDim>>>(work0, wh);
    hipMemcpy(workOffset, &wh, sizeof(uint32_t), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    
    double end = CycleTimer::currentSeconds();
    fprintf(stderr, "Initialized CA in %lf ms\n", (end - ref)*1000.0);
    
    return true;
}

void CASim::getCells(uint8_t *cells) {
    // copy current state from appropriate buffer
    // for now simple 1:1 mapping, each byte being one cell but may change in
    // the future to allow further optimization
    uint8_t *src = (generation & 1) ? cell1 : cell0;
    
    hipMemcpy2D(cells, width, src+pitch, pitch,
                 width, height, hipMemcpyDeviceToHost);
}
